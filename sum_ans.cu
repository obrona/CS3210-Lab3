
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <algorithm>
#include <iostream>
#include <vector>
#include <random>
#include <chrono>
#include <tuple>
#include <utility>
#include <numeric>
#include <iomanip>

// Declare a GPU-visible unsigned long long variable in global memory.
__device__ unsigned long long dResult;

/*
   The simplest reduction operation: every threads atomically adds to the global value.
   atomicAdd is a CUDA built-in function
*/
__global__ void reduceAtomicGlobal(const unsigned long long* __restrict input, unsigned long long N)
{
    const unsigned long long id = threadIdx.x + blockIdx.x * blockDim.x;
    /* 
    Since all blocks must have the same number of threads,
    we may have to launch more threads than there are 
    inputs. Superfluous threads should not try to read 
    from the input (out of bounds access!)
    */
    if (id < N)
        atomicAdd(&dResult, input[id]);
}

/*
   Suggested improvement #1: use shared memory. 
  
   Try to use a __shared__ variable (shared only within the block) to accumulate updates from each thread block.
   Only update the global variable (dResult) at the end of each block's run.
*/
__global__ void reduceAtomicShared(const unsigned long long* __restrict input, unsigned long long N)
{
    const unsigned long long id = threadIdx.x + blockIdx.x * blockDim.x;

    // Declare a shared var for each block
    __shared__ unsigned long long x;

    // Only one thread should initialize this shared value
    if (threadIdx.x == 0) 
        x = 0.0f;
    
    /*
    Before we continue, we must ensure that all threads
    can see this update (initialization) by thread 0
    */
    __syncthreads();

    /*
    Every thread in the block adds its input to the
    shared variable of the block.
    */
    if (id < N) 
        atomicAdd(&x, input[id]);

    // Wait until all threads have done their part
    __syncthreads();

    /*
    Once they are all done, only one thread must add
    the block's partial result to the global variable. 
    */
    if (threadIdx.x == 0) 
        atomicAdd(&dResult, x);
}

/*
 Suggested improvement #2 (May be challenging, but doable!): Using a better algorithm.
 
 Notice that in improvement #1, all threads in a block now contend for the shared variable.
 While this is better than a global variable, it is still not ideal.

 We can use a much better parallel algorithm to reduce contention. 
 See the idea from Lecture 1, Slide 28 ("Better Parallel Algorithm")

 Consider using a shared __array__ of long long values, one for each thread in the block.
 We can compute the parallel result over multiple iterations 
 	(since compute instructions are much faster than memory accesses).

 In each iteration, each thread should accumulate a partial result from the previous iteration. 
 	Not all threads have to be involved in each iteration, the number of threads involved will reduce each time.
 This will make the most sense if you see the appropriate slide in the lecture notes 
 	Instead of each node being a core, now each node is a GPU thread.

 Similarly, in the end, some thread in the block should atomically update dResult

 With this improvement, you should get an algorithm that is closer to O(log N) instead of O(N).

*/
template <int BLOCK_SIZE>
__global__ void reduceShared(const unsigned long long* __restrict input, unsigned long long N)
{
    const unsigned long long id = threadIdx.x + blockIdx.x * blockDim.x;

    /*
       One shared variable for each thread to accumulate a partial sum with less contention.
    */
    __shared__ unsigned long long data[BLOCK_SIZE];

    /*
    Use a new strategy to handle superfluous threads.
    To make sure they stay alive and can help with
    the reduction, threads without an input simply
    produce a '0', which has no effect on the result.
    */
    data[threadIdx.x] = (id < N ? input[id] : 0);

    /*
    log N iterations to complete. In each step, a thread
    accumulates two partial values to form the input for
    the next iteration. The sum of all partial results 
    eventually yields the full result of the reduction. 
    */
    for (unsigned long long s = blockDim.x / 2; s > 0; s /= 2)
    {
        /*
        In each iteration, we must make sure that all
        threads are done writing the updates of the
        previous iteration / the initialization.
        */
        __syncthreads();
        if (threadIdx.x < s)
            data[threadIdx.x] += data[threadIdx.x + s];
    }

    /*
    Note: thread 0 is the last thread to combine two
    partial results, and the one who writes to global
    memory, therefore no synchronization is required
    after the last iteration.
    */
    if (threadIdx.x == 0)
        atomicAdd(&dResult, data[0]);
}

// You do not need to change this
__host__ void prepareArrayCPUGPU(unsigned long long N, std::vector<unsigned long long>& vals, unsigned long long** dValsPtr)
{
    constexpr unsigned long long target = 42;
    std::cout << "\nExpected value: " << target * N << "\n" << std::endl;

    // Generate
    vals.resize(N);
    // There are better ways to do this but this allows for random numbers in the future 
    std::for_each(vals.begin(), vals.end(), [](unsigned long long& f) { f = target; });

    // Allocate some global GPU memory to write the inputs to
    hipMalloc((void**)dValsPtr, sizeof(unsigned long long) * N);
    // Expliclity copy the inputs from the CPU to the GPU
    hipMemcpy(*dValsPtr, vals.data(), sizeof(unsigned long long) * N, hipMemcpyHostToDevice);
}

int main()
{
    /*
     Expected output: Accumulated results from CPU and GPU that equals 42 * NUM_ITEMS 
    */

    constexpr unsigned long long BLOCK_SIZE = 256;
    constexpr unsigned long long WARMUP_ITERATIONS = 10;
    constexpr unsigned long long TIMING_ITERATIONS = 20;
    constexpr unsigned long long N = 10'000'000;

    // Create input arrays in CPU and GPU
    std::cout << "Producing input array...\n\n";
    std::vector<unsigned long long> vals;
    unsigned long long* dValsPtr;
    prepareArrayCPUGPU(N, vals, &dValsPtr);

    std::cout << "==== CPU Reduction ====\n" << std::endl;
    // A reference value is computed by sequential reduction
    unsigned long long referenceResult = std::accumulate(vals.cbegin(), vals.cend(), 0ll);
    std::cout << "Computed CPU value: " << referenceResult << std::endl;

    std::cout << "\n==== GPU Reductions ====\n" << std::endl;
    /*
     Set up a collection of reductions to evaluate for performance. 
     Each entry gives a technique's name, the kernel to call, and
     the number of threads required for each individual technique.
    */
    const std::tuple<const char*, void(*)(const unsigned long long*, unsigned long long), unsigned long long> reductionTechniques[]
    {
        {"Atomic Global", reduceAtomicGlobal, N},
        {"Atomic Shared", reduceAtomicShared, N},
        {"Reduce Shared", reduceShared<BLOCK_SIZE>, N},
	// TODO: add any new functions you want to test here
    };

    // Evaluate each technique separately
    for (const auto& [name, func, numThreads] : reductionTechniques)
    {
        // Compute the smallest grid to start required threads with a given block size
        const dim3 blockDim = { BLOCK_SIZE, 1, 1 };
        const dim3 gridDim = { (numThreads + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1 };

        // Run several reductions for GPU to warm up
        for (unsigned long long i = 0; i < WARMUP_ITERATIONS; i++)
            func<<<gridDim, blockDim>>>(dValsPtr, N);

        // Synchronize to ensure CPU only records time after warmup is done
        hipDeviceSynchronize();
        const auto before = std::chrono::system_clock::now();

        unsigned long long result = 0.0f;
        // Run several iterations to get an average measurement
        for (unsigned long long i = 0; i < TIMING_ITERATIONS; i++)
        {
            // Reset acummulated result to 0 in each run
            hipMemcpyToSymbol(HIP_SYMBOL(dResult), &result, sizeof(unsigned long long));
            func<<<gridDim, blockDim>>>(dValsPtr, N);
        }

        // cudaMemcpyFromSymbol will implicitly synchronize CPU and GPU
        hipMemcpyFromSymbol(&result, HIP_SYMBOL(dResult), sizeof(unsigned long long));

        // Can measure time without an extra synchronization
        const auto after = std::chrono::system_clock::now();
        const auto elapsed = 1000.f * std::chrono::duration_cast<std::chrono::duration<float>>(after - before).count();
	const auto status = result == referenceResult ? "OK" : "FAILED";
        std::cout << std::setw(20) << name << "\t" << elapsed / TIMING_ITERATIONS << "ms \t" << std::setw(10) << result  << "\t" << status << std::endl;
    }

    // Free the allocated memory for input
    hipFree(dValsPtr);
    return 0;
}

